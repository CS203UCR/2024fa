#include "hip/hip_runtime.h"
/*
 *  file name: matrix.cu
 *
 *  matrix.cu contains the code that realize some common used matrix operations in CUDA
 *  
 *  this is a toy program for learning CUDA, some functions are reusable in other project
 *  
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16

#ifndef D_TYPE
#define D_TYPE float
#endif
double getCurrentTimestamp();
int ARRAY_SIZE;
/*
*********************************************************************
function name: gpu_matrix_mult

description: dot product of two matrix (not only square)

parameters: 
            &a GPU device pointer to a m X n matrix (A)
            &b GPU device pointer to a n X k matrix (B)
            &c GPU device output purpose pointer to a m X k matrix (C) 
            to store the result

Note:
    grid and block should be configured as:
        dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
*********************************************************************
*/
__global__ void gpu_matrix_mult(D_TYPE *a,D_TYPE *b, D_TYPE *c, int size)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    D_TYPE sum = 0;
        for(int i = 0; i < size; i++) 
        {
            sum += a[row * size + i] * b[i * size + col];
        }
        c[row * size + col] = sum;
} 

/*
*********************************************************************
function name: gpu_square_matrix_mult

description: dot product of two matrix (not only square) in GPU

parameters: 
            &a GPU device pointer to a n X n matrix (A)
            &b GPU device pointer to a n X n matrix (B)
            &c GPU device output purpose pointer to a n X n matrix (C) 
            to store the result
Note:
    grid and block should be configured as:

        dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

return: none
*********************************************************************
*/
__global__ void gpu_block_matrix_mult(D_TYPE *d_a, D_TYPE *d_b, D_TYPE *d_result, int n) 
{
    __shared__ D_TYPE tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ D_TYPE tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    D_TYPE tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

/*
*********************************************************************
function name: gpu_matrix_transpose

description: matrix transpose

parameters: 
            &mat_in GPU device pointer to a rows X cols matrix
            &mat_out GPU device output purpose pointer to a cols X rows matrix 
            to store the result
Note:
    grid and block should be configured as:
        dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

return: none
*********************************************************************
*/
__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols) 
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows) 
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}
/*
*********************************************************************
function name: cpu_matrix_mult

description: dot product of two matrix (not only square) in CPU, 
             for validating GPU results

parameters: 
            &a CPU host pointer to a m X n matrix (A)
            &b CPU host pointer to a n X k matrix (B)
            &c CPU host output purpose pointer to a m X k matrix (C) 
            to store the result
return: none
*********************************************************************
*/
void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

/*
*********************************************************************
function name: main

description: test and compare

parameters: 
            none

return: none
*********************************************************************
*/
int main(int argc, char const *argv[])
{
    int size;
    /* Fixed seed for illustration */
  D_TYPE *a, *b, *c;
  int mode,i;
  if(argc >= 2)
    ARRAY_SIZE = atoi(argv[1]);
  mode = atoi(argv[2]);
  printf("Data Type Size: %d\n", sizeof(D_TYPE));
  a = (D_TYPE *)malloc(ARRAY_SIZE*ARRAY_SIZE*sizeof(D_TYPE));
  if(a == NULL)
   fprintf(stderr,"allocating array a failed\n");
  for(i = 0; i < ARRAY_SIZE*ARRAY_SIZE; i++)
      a[i] = (D_TYPE)rand();

  b = (D_TYPE *)malloc(ARRAY_SIZE*ARRAY_SIZE*sizeof(D_TYPE));
  if(b == NULL)
   fprintf(stderr,"allocating array b failed\n");

  for(i = 0; i < ARRAY_SIZE*ARRAY_SIZE; i++)
      b[i] = (D_TYPE)rand();

  c = (D_TYPE *)malloc(ARRAY_SIZE*ARRAY_SIZE*sizeof(D_TYPE));
  if(c == NULL)
   fprintf(stderr,"allocating array c failed\n");
    double start_time = getCurrentTimestamp();
    // allocate memory in host RAM, h_cc is used to store CPU result

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
    // Allocate memory space on the device 
    D_TYPE *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(D_TYPE)*ARRAY_SIZE*ARRAY_SIZE);
    hipMalloc((void **) &d_b, sizeof(D_TYPE)*ARRAY_SIZE*ARRAY_SIZE);
    hipMalloc((void **) &d_c, sizeof(D_TYPE)*ARRAY_SIZE*ARRAY_SIZE);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, a, sizeof(D_TYPE)*ARRAY_SIZE*ARRAY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(D_TYPE)*ARRAY_SIZE*ARRAY_SIZE, hipMemcpyHostToDevice);

    unsigned int grid_rows = (ARRAY_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (ARRAY_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // Launch kernel 
    if(mode)
    gpu_block_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, ARRAY_SIZE);    
    else
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, ARRAY_SIZE);    
    // Transefr results from device to host 
    hipMemcpy(c, d_c, sizeof(D_TYPE)*ARRAY_SIZE*ARRAY_SIZE, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    double end_time = getCurrentTimestamp();
    double total_time = end_time - start_time;
    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", ARRAY_SIZE, ARRAY_SIZE, ARRAY_SIZE, ARRAY_SIZE, gpu_elapsed_time_ms);
    const float flops = (float)(2.0f * ARRAY_SIZE * ARRAY_SIZE * ARRAY_SIZE/(gpu_elapsed_time_ms/1e3));
    printf("\nThroughput: %0.2f GFLOPS\n\n", flops * 1e-9);
    fprintf(stderr,"%ld, %0.2f, %f, %lf\n", ARRAY_SIZE, flops * 1e-9, gpu_elapsed_time_ms, total_time);

/*
    // validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            //printf("[%d][%d]:%d == [%d][%d]:%d, ", i, j, h_c[i*k + j], i, j, h_c[i*k + j]);
            if(h_c[i*k + j] != h_c[i*k + j])
            {
                all_ok = 0;
            }
        }
        //printf("\n");
    }

    // roughly compute speedup
    if(all_ok)
    {
        printf("all results are correct!!!, speedup = %f\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    }
    else
    {
        printf("incorrect results\n");
    }
*/
    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}

double getCurrentTimestamp() {
#ifdef _WIN32 // Windows
  // Use the high-resolution performance counter.

  static LARGE_INTEGER ticks_per_second = {};
  if(ticks_per_second.QuadPart == 0) {
    // First call - get the frequency.
    QueryPerformanceFrequency(&ticks_per_second);
  }

  LARGE_INTEGER counter;
  QueryPerformanceCounter(&counter);

  double seconds = double(counter.QuadPart) / double(ticks_per_second.QuadPart);
  return seconds;
#else         // Linux
  timespec a;
  clock_gettime(CLOCK_MONOTONIC, &a);
  return (double(a.tv_nsec) * 1.0e-9) + double(a.tv_sec);
#endif
}
